#include "hip/hip_runtime.h"
#include "book.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>

__global__ void doSomeComputation(double *a) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    double result = 0;
    for (int i = 0; i < 10000; i++) {
        result += sqrt(pow(3.14159,-i));
    }
    a[tid] = result;
}

int main(int argc, char *argv[]) {
    int blockNum = atoi(argv[1]);
    int threadNum = atoi(argv[2]);

    double a[blockNum * threadNum];
    double *dev_a;
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, blockNum * threadNum * sizeof(double) ) );

    doSomeComputation<<<blockNum, threadNum>>>(dev_a);

    HANDLE_ERROR(hipMemcpy(a, dev_a, blockNum * threadNum * sizeof(double), hipMemcpyDeviceToHost));

    // for (int i = 0; i < blockNum * threadNum; i++) {
    //     printf("a[%d] = %f\n", i, a[i]);
    // }

    hipFree(dev_a);

    return 0;
}