#include "hip/hip_runtime.h"
#include "book.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>

__global__ void matrixMul(double *a. double *b, double *c, int p) {
    int tid = blockIdx.x * gridDim.x + blockIdx.y;
    for (int i = 0; i < p; i++) {
        c[tid] = c[tid] + a[blockIdx.x * p + i] * b[i * gridDim.y + blockIdx.y];
    }
}

int main(int argc, char *argv[]) {
    int m = atoi(argv[1]);
    int p = atoi(argv[2]);
    int n = atoi(argv[3]);

    double a[m * p];
    double b[p * n];
    double c[m * n];
    double *dev_a, *dev_b, dev_c;

    for (int i = 0; i < m; i++) {
        for (int j = 0; j < p; j++) {
            a[i * p + j] = 1;
        }
    }

    for (int i = 0; i < p; i++) {
        for (int j = 0; j < n; j++) {
            b[i * n + j] = 1;
        }
    }

    HANDLE_ERROR( hipMalloc( (void**)&dev_a, m * p * sizeof(double) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, p * n * sizeof(double) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, m * n * sizeof(double) ) );

    HANDLE_ERROR(hipMemcpy(dev_a, a, m * p * sizeof(double), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, p * n * sizeof(double), hipMemcpyHostToDevice));

    dim3 grid(m, n);
    matrixMul<<<grid, 1>>>(dev_a, dev_b, dev_c, p);

    HANDLE_ERROR(hipMemcpy(c, dev_c, m * n * sizeof(double), hipMemcpyDeviceToHost));

    // for (int i = 0; i < m; i++) {
    //     for (int j = 0; j < n; j++) {
    //         printf("c[%d][%d] = %f\n", i, j, c[i * n + j]);
    //     }
    // }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}