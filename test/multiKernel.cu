#include "hip/hip_runtime.h"
#include "book.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>

__global__ void doSomeComputation(double *a) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    double result = 0;
    for (int i = 0; i < 10000; i++) {
        result += sqrt(pow(3.14159,-i));
    }
    a[tid] = result;
}

int main(int argc, char *argv[]) {
    int blockNum = atoi(argv[1]);
    int threadNum = atoi(argv[2]);
    int kernelNum = atoi(argv[3]);

    double a[kernelNum][blockNum * threadNum];
    double *dev_a[kernelNum];
    hipStream_t streams[kernelNum];

    for(int i = 0; i < kernelNum; i++) {
        HANDLE_ERROR( hipMalloc( (void**)&dev_a[i], blockNum * threadNum * sizeof(double) ) );
    }
    for(int i = 0; i < kernelNum; i++) {
        hipStreamCreate(&streams[i]);
    }
    for(int i = 0; i < kernelNum; i++) {
        doSomeComputation<<<blockNum, threadNum, 0, streams[i]>>>(dev_a[i]);
    }   
    
    for(int i = 0; i < kernelNum; i++) {
        HANDLE_ERROR(hipMemcpy(a[i], dev_a[i], blockNum * threadNum * sizeof(double), hipMemcpyDeviceToHost));
    } 

    // for (int i = 0; i < kernelNum; i++) {
    //     printf("kernel %d:\n", i);
    //     for (int j = 0; j < blockNum * threadNum; j++) {
    //         printf("a[%d] = %f\n", j, a[j]);
    //     }
    //     printf("\n");
    // }
   
    for(int i = 0; i < kernelNum; i++) {
        hipFree(dev_a[i]);
    } 
    
    return 0;
}