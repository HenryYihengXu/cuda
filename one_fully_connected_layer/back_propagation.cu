#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t backPropagation(double *out, double *x, 
    double *y, double *W, unsigned int column, unsigned int row, double eta);

__global__ void subtractKernel(double *out, double *y)
{
    int i = threadIdx.x;
    out[i] -= y[i];
}

__global__ void updateWKernel(double *out, double *x, double *W, int column, double eta)
{
    int i = threadIdx.x;
    W[i] = W[i] - eta * out[i / column] * x[i % column];
}


int main()
{
    int row = 3;
    int column = 5;
    double W[row * column] = { 
        1, 2, 3, 4, 5,
        2, 4, 6, 8, 10,
        10, 20, 30, 40, 50
    };
    double x[column] = { 10, 10, 10, 10, 10 };
    double out[column] = {150, 300, 1500};
    double y[row] = { 140, 310, 1600 };
    double eta = 0.01;

    // Add vectors in parallel.
    hipError_t cudaStatus = backPropagation(out, x, y, W, column, row, eta);
    printf("new W = \n{ \n %.4f, %.4f, %.4f, %.4f, %.4f\n %.4f, %.4f, %.4f, %.4f, %.4f\n %.4f, %.4f, %.4f, %.4f, %.4f\n}\n",
        W[0], W[1], W[2], W[3], W[4], W[5], W[6], W[7], W[8], W[9], W[10], W[11], W[12], W[13], W[14]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t backPropagation(double *out, double *x, 
    double *y, double *W, unsigned int column, unsigned int row, double eta)
{
    double *dev_y = 0;
    double *dev_x = 0;
    double *dev_out = 0;
    double *dev_W = 0;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&dev_y, row * sizeof(double));
    cudaStatus = hipMalloc((void**)&dev_x, column * sizeof(double));
    cudaStatus = hipMalloc((void**)&dev_out, row * sizeof(double));
    cudaStatus = hipMalloc((void**)&dev_W, row * column * sizeof(double));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_y, y, row * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_x, x, column * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_out, out, row * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_W, W, row * column * sizeof(double), hipMemcpyHostToDevice);
    
    
    // Compute (out - y) to get the differential of cost on predictions
    subtractKernel<<<1, column>>>(dev_out, dev_y);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Compute the differential of cost on weights and update weights: W = W - eta*(delta*(x)T)
    updateWKernel<<<1, column * row>>>(dev_out, dev_x, dev_W, column, eta);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(W, dev_W, row * column * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_out);
    hipFree(dev_y);
    hipFree(dev_x);
    hipFree(dev_W);
    
    return cudaStatus;
}

