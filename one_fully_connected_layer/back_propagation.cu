#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

hipError_t backPropagation(double *out, double *x, 
    double *y, double *W, unsigned int row, unsigned int column, double eta);

__global__ void subtractKernel(double *out, double *y, unsigned int row)
{
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_idx >= row) {
        return;
    }
    out[thread_idx] -= y[thread_idx];
}

__global__ void updateWKernel(double *out, double *x, double *W, 
    unsigned int row, unsigned int column, double eta)
{
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_idx >= row * column) {
        return;
    }
    W[thread_idx] = W[thread_idx] - eta * out[thread_idx / column] * x[thread_idx % column];
}


int main(int argc, char *argv[])
{
    unsigned int row = atoi(argv[1]);
    unsigned int column = atoi(argv[2]);
    double eta = atof(argv[3]);
    double *W = (double*)malloc(row * column * sizeof(double));
    double *x = (double*)malloc(column * sizeof(double));
    double *y = (double*)malloc(row * sizeof(double));
    double *out = (double*)malloc(row * sizeof(double));

    for (int i = 0; i < column; i++) {
        x[i] = 10;
    }

    for (int i = 0; i < row * column; i++) {
        W[i] = 10;
    }

    for (int i = 0; i < column; i++) {
        y[i] = 10;
    }

    for (int i = 0; i < column; i++) {
        out[i] = 11;
    }

    // Add vectors in parallel.
    hipError_t cudaStatus = backPropagation(out, x, y, W, row, column, eta);
    
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < column; j++) {
            printf("%.2f ", W[i * column + j]);
        }
        printf("\n");
    }
    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t backPropagation(double *out, double *x, 
    double *y, double *W, unsigned int row, unsigned int column, double eta)
{
    double *dev_out = 0;
    double *dev_x = 0;
    double *dev_y = 0;
    double *dev_W = 0;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&dev_out, row * sizeof(double));
    cudaStatus = hipMalloc((void**)&dev_x, column * sizeof(double));
    cudaStatus = hipMalloc((void**)&dev_y, row * sizeof(double));
    cudaStatus = hipMalloc((void**)&dev_W, row * column * sizeof(double));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_out, out, row * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_x, x, column * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_y, y, row * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_W, W, row * column * sizeof(double), hipMemcpyHostToDevice);
    
    // Compute (out - y) to get the differential of cost on predictions
    subtractKernel<<<column / 512 + 1, 512>>>(dev_out, dev_y, row);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Compute the differential of cost on weights and update weights: W = W - eta*(delta*(x)T)
    updateWKernel<<<row * column / 512 + 1, 512>>>(dev_out, dev_x, dev_W, row, column, eta);

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(W, dev_W, row * column * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_out);
    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_W);
    
    return cudaStatus;
}

