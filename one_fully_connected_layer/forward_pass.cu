#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

hipError_t forwardPass(double *x, double *y,
    double *W, int row, int column);

__global__ void vectorMultiplicationKernel(double *x, double *y, double *W,
    int row, int column)
{
    int tid = blockIdx.x;
    if (tid >= row) {
        return;
    }
    double result = 0;
    for (int j = 0; j < column; j++) {
        result += W[tid * column + j] * x[j];
    }
    for (int j = 0; j < 10000; j++) {
        for (int k = 0; k < 10000; k++) {
            result++;
            result--;
        }
    }
    y[tid] = result;
}

int main(int argc, char *argv[])
{
    int row = atoi(argv[1]);
    int column = atoi(argv[2]);

    double *W = (double*)malloc(row * column * sizeof(double));
    double *x = (double*)malloc(column * sizeof(double));
    double *y = (double*)malloc(row * sizeof(double));

    for (int i = 0; i < column; i++) {
        x[i] = 10;
    }

    for (int i = 0; i < row * column; i++) {
        W[i] = 10;
    }

    hipError_t cudaStatus = forwardPass(x, y, W, row, column);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vectorMultiplicationWithCuda failed!");
        return 1;
    }

    for (int i = 0; i < row; i++) {
        printf("%.2f ", y[i]);
    }
    printf("\n");

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t forwardPass(double *x, double *y, double *W,
    int row, int column)
{
    double *dev_x = 0;
    double *dev_y = 0;
    double *dev_W = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_x, column * sizeof(double));
    cudaStatus = hipMalloc((void**)&dev_y, row * sizeof(double));
    cudaStatus = hipMalloc((void**)&dev_W, row * column * sizeof(double));

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_x, x, column * sizeof(double), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_W, W, row * column * sizeof(double), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    vectorMultiplicationKernel<<<row, 1>>>(dev_x, dev_y, dev_W, row, column);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vectorMultiplicationKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(y, dev_y, row * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_W);

    return cudaStatus;
}
