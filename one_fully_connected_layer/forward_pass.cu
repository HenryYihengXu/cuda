#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t vectorMatrixMulWithCuda(int *c, const int *a, 
    const int *b, unsigned int column, unsigned int row);

__global__ void vectorMultiplicationKernel(int *c, const int *a, const int *b, unsigned int column)
{
    int i = threadIdx.x;
    int result = 0;
    for (int j = 0; j < column; j++) {
        result += a[i * column + j] * b[j];
    }
    c[i] = result;
}

int main()
{
    const int row = 3;
    const int column = 5;
    const int a[row * column] = { 
        1, 2, 3, 4, 5,
        2, 4, 6, 8, 10,
        10, 20, 30, 40, 50
    };
    const int b[column] = { 10, 10, 10, 10, 10 };
    int c[row] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = vectorMatrixMulWithCuda(c, a, b, column, row);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vectorMultiplicationWithCuda failed!");
        return 1;
    }

    printf("{\n 1,  2,  3,  4,  5,\n 2,  4,  6,  8,  10,\n 10, 20, 30, 40, 50\n} * \n{10, 10, 10, 10, 10}\n = {%d, %d, %d}\n",
        c[0], c[1], c[2]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t vectorMatrixMulWithCuda(int *c, const int *a, const int *b, unsigned int column, unsigned int row)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, row * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, row * column * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, column * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, row * column * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, column * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    vectorMultiplicationKernel<<<1, row>>>(dev_c, dev_a, dev_b, column);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, row * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}