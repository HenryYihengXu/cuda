#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t vectorMatrixMulWithCuda(double *c, const double *a, 
    const double *b, unsigned int column, unsigned int row);

__global__ void vectorMultiplicationKernel(double *c, const double *a, const double *b, unsigned int column)
{
    int i = threadIdx.x;
    double result = 0;
    for (int j = 0; j < column; j++) {
        result += a[i * column + j] * b[j];
    }
    c[i] = result;
}

int main()
{
    const int row = 3;
    const int column = 5;
    const double W[row * column] = { 
        1, 2, 3, 4, 5,
        2, 4, 6, 8, 10,
        10, 20, 30, 40, 50
    };
    const double x[column] = { 10, 10, 10, 10, 10 };
    double y[row] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = vectorMatrixMulWithCuda(y, W, x, column, row);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vectorMultiplicationWithCuda failed!");
        return 1;
    }

    printf("{\n 1,  2,  3,  4,  5,\n 2,  4,  6,  8,  10,\n 10, 20, 30, 40, 50\n} * \n{10, 10, 10, 10, 10}\n = {%0.4f, %0.4f, %0.4f}\n",
        y[0], y[1], y[2]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t vectorMatrixMulWithCuda(double *c, const double *a, const double *b, unsigned int column, unsigned int row)
{
    double *dev_a = 0;
    double *dev_b = 0;
    double *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, row * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, row * column * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, column * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, row * column * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, column * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    vectorMultiplicationKernel<<<1, row>>>(dev_c, dev_a, dev_b, column);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, row * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

