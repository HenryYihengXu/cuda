#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

hipError_t forwardPass(double *x,  double *y, 
     double *W, int row, int column);

__global__ void addKernel(double *y, double *W, 
    int row, int column)
{
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_idx >= row) {
        return;
    }
    double result = 0;
    for (int j = 0; j < column; j++) {
        result += W[thread_idx * column + j];
    }
    y[thread_idx] = result;
}

__global__ void mulKernel(double *x,  double *W, 
    int row, int column)
{
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_idx >= row * column) {
        return;
    }
    W[thread_idx] = W[thread_idx] * x[thread_idx / column];
}

int main(int argc, char *argv[])
{
    int row = atoi(argv[1]);
    int column = atoi(argv[2]);

    double *W = (double*)malloc(row * column * sizeof(double));
    double *x = (double*)malloc(column * sizeof(double));
    double *y = (double*)malloc(row * sizeof(double));

    for (int i = 0; i < column; i++) {
        x[i] = 10;
    }

    for (int i = 0; i < row * column; i++) {
        W[i] = 10;
    }

    hipError_t cudaStatus = forwardPass(x, y, W, row, column);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vectorMultiplicationWithCuda failed!");
        return 1;
    }

    for (int i = 0; i < row; i++) {
        printf("%.2f ", y[i]);
    }
    printf("\n");

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t forwardPass(double *x,  double *y,  double *W, 
    int row, int column)
{
    double *dev_x = 0;
    double *dev_y = 0;
    double *dev_W = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_x, column * sizeof(double));
    cudaStatus = hipMalloc((void**)&dev_y, row * sizeof(double));
    cudaStatus = hipMalloc((void**)&dev_W, row * column * sizeof(double));

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_x, x, column * sizeof(double), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_W, W, row * column * sizeof(double), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    mulKernel<<<row * column / 512 + 1, 512>>>(dev_x, dev_W, row, column);
    //mulKernel<<<1, row * column>>>(dev_x, dev_W, row, column);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "mulKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

     // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<row / 512 + 1, 512>>>(dev_y, dev_W, row, column);
    //addKernel<<<1, row>>>(dev_y, dev_W, row, column);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(y, dev_y, row * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_x);
    hipFree(dev_y);
    hipFree(dev_W);
    
    return cudaStatus;
}



