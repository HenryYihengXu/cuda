#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

hipError_t vectorMatrixMulWithCuda(double *c,  double *a, 
     double *b, unsigned int column, unsigned int row);
    
void size_3x5_test();
void size_10x10_test();

__global__ void addKernel(double *d, double *c, unsigned int column)
{
    int i = threadIdx.x;
    double result = 0;
    for (int j = 0; j < column; j++) {
        result += c[i * column + j];
    }
    d[i] = result;
}

__global__ void mulKernel(double *c,  double *a,  double *b, unsigned int column)
{
    int i = threadIdx.x;
    c[i] = a[i] * b[i / column];
}

int main()
{
    size_3x5_test();
    size_10x10_test();

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t vectorMatrixMulWithCuda(double *d,  double *a,  double *b, unsigned int column, unsigned int row)
{
    double *dev_a = 0;
    double *dev_b = 0;
    double *dev_c = 0;
    double *dev_d = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, row * column * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, row * column * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, column * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_d, row * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, row * column * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, column * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    mulKernel<<<1, row * column>>>(dev_c, dev_a, dev_b, column);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "mulKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

     // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, row>>>(dev_d, dev_c, column);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(d, dev_d, row * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_d);
    
    return cudaStatus;
}

void size_3x5_test() {
    int row = 3;
    int column = 5;
    double W[row * column] = { 
        1, 2, 3, 4, 5,
        2, 4, 6, 8, 10,
        10, 20, 30, 40, 50
    };
    double x[column] = { 10, 10, 10, 10, 10 };
    double y[row] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = vectorMatrixMulWithCuda(y, W, x, column, row);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vectorMultiplicationWithCuda failed!");
        return;
    }

    printf("{\n 1,  2,  3,  4,  5,\n 2,  4,  6,  8,  10,\n 10, 20, 30, 40, 50\n} * \n{10, 10, 10, 10, 10}\n = {%.2f, %.2f, %.2f}\n",
        y[0], y[1], y[2]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return;
    }
}

void size_10x10_test() {
    int row = 10;
    int column = 10;
    double W[row * column] = {0};
    double x[column] = {0};
    double y[row] = { 0 };

    for (int i = 0; i < row * column; i++) {
        W[i] = 10;
    }

    for (int i = 0; i < column; i++) {
        x[i] = 10;
    }

    // Add vectors in parallel.
    hipError_t cudaStatus = vectorMatrixMulWithCuda(y, W, x, column, row);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vectorMultiplicationWithCuda failed!");
        return;
    }

    for (int i = 0; i < row; i++) {
        printf("%.2f ", y[i]);
    }
    printf("\n");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return;
    }
}


