#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t vectorMatrixMulWithCuda(int *c, const int *a, 
    const int *b, unsigned int column, unsigned int row);

__global__ void addKernel(int *d, int *c, unsigned int column)
{
    int i = threadIdx.x;
    int result = 0;
    for (int j = 0; j < column; j++) {
        result += c[i * column + j];
    }
    d[i] = result;
}

__global__ void mulKernel(int *c, const int *a, const int *b, unsigned int column)
{
    int i = threadIdx.x;
    c[i] = a[i] * b[i / column];
}

int main()
{
    const int row = 3;
    const int column = 5;
    const int a[row * column] = { 
        1, 2, 3, 4, 5,
        2, 4, 6, 8, 10,
        10, 20, 30, 40, 50
    };
    const int b[column] = { 10, 10, 10, 10, 10 };
    int d[row] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = vectorMatrixMulWithCuda(d, a, b, column, row);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vectorMultiplicationWithCuda failed!");
        return 1;
    }

    printf("{\n 1,  2,  3,  4,  5,\n 2,  4,  6,  8,  10,\n 10, 20, 30, 40, 50\n} * \n{10, 10, 10, 10, 10}\n = {%d, %d, %d}\n",
        d[0], d[1], d[2]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t vectorMatrixMulWithCuda(int *d, const int *a, const int *b, unsigned int column, unsigned int row)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    int *dev_d = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, row * column * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, row * column * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, column * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_d, row * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, row * column * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, column * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    mulKernel<<<1, row * column>>>(dev_c, dev_a, dev_b, column);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

     // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, row>>>(dev_d, dev_c, column);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(d, dev_d, row * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_d);
    
    return cudaStatus;
}
