#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

hipError_t forwardPass(double *x1, double *y1, double *W1, 
    double *x2, double *y2, double *W2, 
    int row, int column);

__global__ void vectorMultiplicationKernel(double *x, double *y, double *W,
    int row, int column)
{
    int tid = blockIdx.x;
    if (tid >= row) {
        return;
    }
    double result = 0;
    for (int j = 0; j < column; j++) {
        result += W[tid * column + j] * x[j];
    }
    for (int j = 0; j < 10000; j++) {
        for (int k = 0; k < 10000; k++) {
            result++;
            result--;
        }
    }
    y[tid] = result;
}

int main(int argc, char *argv[])
{
    int row = atoi(argv[1]);
    int column = atoi(argv[2]);

    double *W1 = (double*)malloc(row * column * sizeof(double));
    double *x1 = (double*)malloc(column * sizeof(double));
    double *y1 = (double*)malloc(row * sizeof(double));
    double *W2 = (double*)malloc(row * column * sizeof(double));
    double *x2 = (double*)malloc(column * sizeof(double));
    double *y2 = (double*)malloc(row * sizeof(double));

    for (int i = 0; i < column; i++) {
        x1[i] = 10;
    }

    for (int i = 0; i < row * column; i++) {
        W1[i] = 10;
    }

    for (int i = 0; i < column; i++) {
        x2[i] = 10;
    }

    for (int i = 0; i < row * column; i++) {
        W2[i] = 10;
    }

    hipError_t cudaStatus = forwardPass(x1, y1, W1, x2, y2, W2, row, column);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vectorMultiplicationWithCuda failed!");
        return 1;
    }

    for (int i = 0; i < row; i++) {
        printf("%.2f ", y1[i]);
    }
    printf("\n");

    for (int i = 0; i < row; i++) {
        printf("%.2f ", y2[i]);
    }
    printf("\n");

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t forwardPass(double *x1, double *y1, double *W1,
    double *x2, double *y2, double *W2,
    int row, int column)
{
    double *dev_x1 = 0;
    double *dev_y1 = 0;
    double *dev_W1 = 0;
    double *dev_x2 = 0;
    double *dev_y2 = 0;
    double *dev_W2 = 0;
    hipError_t cudaStatus;

    // Allocate GPU buffers for three vectors (two input, one output)    .
    
    hipMalloc((void**)&dev_x1, column * sizeof(double));
    hipMalloc((void**)&dev_y1, row * sizeof(double));
    hipMalloc((void**)&dev_W1, row * column * sizeof(double));
    hipMemcpy(dev_x1, x1, column * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_W1, W1, row * column * sizeof(double), hipMemcpyHostToDevice);

    hipMalloc((void**)&dev_x2, column * sizeof(double));
    hipMalloc((void**)&dev_y2, row * sizeof(double));
    hipMalloc((void**)&dev_W2, row * column * sizeof(double));
    hipMemcpy(dev_x2, x2, column * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_W2, W2, row * column * sizeof(double), hipMemcpyHostToDevice);

    //Launch a kernel on the GPU with one thread for each element.
    hipStream_t s1;
    hipStream_t s2;
    hipStreamCreate(&s1);
    vectorMultiplicationKernel<<<row, 1, 0, s1>>>(dev_x1, dev_y1, dev_W1, row, column);

    hipStreamCreate(&s2);
    vectorMultiplicationKernel<<<row, 1, 0, s2>>>(dev_x2, dev_y2, dev_W2, row, column);
    // vectorMultiplicationKernel<<<row, 1>>>(dev_x1, dev_y1, dev_W1, row, column);
    // vectorMultiplicationKernel<<<row, 1>>>(dev_x2, dev_y2, dev_W2, row, column);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(y1, dev_y1, row * sizeof(double), hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(y2, dev_y2, row * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(dev_x1);
    hipFree(dev_y1);
    hipFree(dev_W1);
    hipFree(dev_x2);
    hipFree(dev_y2);
    hipFree(dev_W2);

    return cudaStatus;
}


