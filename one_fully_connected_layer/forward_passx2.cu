#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

hipError_t forwardPass(double *x1, double *y1, double *W1, 
    double *x2, double *y2, double *W2, 
    int row, int column);

__global__ void vectorMultiplicationKernel(double *x, double *y, double *W,
    int row, int column)
{
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_idx >= row) {
        return;
    }
    double result = 0;
    for (int j = 0; j < column; j++) {
        result += W[thread_idx * column + j] * x[j];
    }
    y[thread_idx] = result;
}

int main(int argc, char *argv[])
{
    int row = atoi(argv[1]);
    int column = atoi(argv[2]);

    double *W1 = (double*)malloc(row * column * sizeof(double));
    double *x1 = (double*)malloc(column * sizeof(double));
    double *y1 = (double*)malloc(row * sizeof(double));
    double *W2 = (double*)malloc(row * column * sizeof(double));
    double *x2 = (double*)malloc(column * sizeof(double));
    double *y2 = (double*)malloc(row * sizeof(double));

    for (int i = 0; i < column; i++) {
        x1[i] = 10;
    }

    for (int i = 0; i < row * column; i++) {
        W1[i] = 10;
    }

    for (int i = 0; i < column; i++) {
        x2[i] = 10;
    }

    for (int i = 0; i < row * column; i++) {
        W2[i] = 10;
    }

    hipError_t cudaStatus = forwardPass(x1, y1, W1, x2, y2, W2, row, column);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vectorMultiplicationWithCuda failed!");
        return 1;
    }

    for (int i = 0; i < row; i++) {
        printf("%.2f ", y1[i]);
    }
    printf("\n");

    for (int i = 0; i < row; i++) {
        printf("%.2f ", y2[i]);
    }
    printf("\n");

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t forwardPass(double *x1, double *y1, double *W1,
    double *x2, double *y2, double *W2,
    int row, int column)
{
    double *dev_x1 = 0;
    double *dev_y1 = 0;
    double *dev_W1 = 0;
    double *dev_x2 = 0;
    double *dev_y2 = 0;
    double *dev_W2 = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_x1, column * sizeof(double));
    cudaStatus = hipMalloc((void**)&dev_y1, row * sizeof(double));
    cudaStatus = hipMalloc((void**)&dev_W1, row * column * sizeof(double));
    cudaStatus = hipMalloc((void**)&dev_x2, column * sizeof(double));
    cudaStatus = hipMalloc((void**)&dev_y2, row * sizeof(double));
    cudaStatus = hipMalloc((void**)&dev_W2, row * column * sizeof(double));

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_x1, x1, column * sizeof(double), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_W1, W1, row * column * sizeof(double), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_x2, x2, column * sizeof(double), hipMemcpyHostToDevice);
    cudaStatus = hipMemcpy(dev_W2, W2, row * column * sizeof(double), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    vectorMultiplicationKernel<<<row / 512 + 1, 512>>>(dev_x1, dev_y1, dev_W1, row, column);
    vectorMultiplicationKernel<<<row / 512 + 1, 512>>>(dev_x2, dev_y2, dev_W2, row, column);
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vectorMultiplicationKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(y1, dev_y1, row * sizeof(double), hipMemcpyDeviceToHost);
    cudaStatus = hipMemcpy(y2, dev_y2, row * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_x1);
    hipFree(dev_y1);
    hipFree(dev_W1);
    hipFree(dev_x2);
    hipFree(dev_y2);
    hipFree(dev_W2);

    return cudaStatus;
}

